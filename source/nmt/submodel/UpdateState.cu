#include "hip/hip_runtime.h"
#include "UpdateState.cuh"

namespace nmt {

  // src (N, B, L, H)
  // tgt (N, B, L, H)
  // index (B)
  __global__ void updateStateKernel(const float* const src,
                                    const int* const index,
                                    const UpdateStateParams params,
                                    float * const tgt) {
    // niutensor doesn't guarantee 128-bit aligned.
    // TODO(umiswing): Use 128-bit load after 128-bit aligned support.
    constexpr int ELEMENTS_PER_THREAD = 1;
    const int THREADS_PER_BLOCK = blockDim.x;
    const int ELEMENTS_PER_BLOCK = ELEMENTS_PER_THREAD * THREADS_PER_BLOCK;
  
    const int src_hid = blockIdx.y;
    const int src_bid = index[blockIdx.x];
    const int src_hbid = (src_hid * params.src_batch_size + src_bid) * params.seqlen * params.head_dim;
  
    const int tgt_hid = blockIdx.y;
    const int tgt_bid = blockIdx.x;
    const int tgt_hbid = (tgt_hid * params.tgt_batch_size + tgt_bid) * params.seqlen * params.head_dim;
    #pragma unroll
    for(int i=threadIdx.x*ELEMENTS_PER_THREAD;i<params.seqlen * params.head_dim / ELEMENTS_PER_THREAD;i+=ELEMENTS_PER_BLOCK) {
      tgt[tgt_hbid+i] = src[src_hbid+i];
    }
  }
  
  void updateState(XTensor *s, XTensor* index, struct UpdateStateParams params, XTensor* t) {
    int devID = s->devID;
    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    dim3 blocks(params.tgt_batch_size, params.num_head);

    dim3 threads(params.seqlen*params.head_dim > GDevs.GPUs[devID].GPUMaxThreadNumPerBlock ?
                 GDevs.GPUs[devID].GPUMaxThreadNumPerBlock :
                 params.seqlen*params.head_dim );

    updateStateKernel<<<blocks, threads>>>(static_cast<float*>(s->data),
                                           static_cast<int*>(index->data),
                                           params,
                                           static_cast<float*>(t->data));

    BacktoCudaDev(devID, devIDBackup);
  }
}