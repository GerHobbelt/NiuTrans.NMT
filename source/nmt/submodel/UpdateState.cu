#include "hip/hip_runtime.h"
/* NiuTrans.NMT - an open-source neural machine translation system.
 * Copyright (C) 2024 NiuTrans Research. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: umiswing (umiswing@foxmail.com) 2024-03
 */

#include "UpdateState.cuh"

namespace nmt {

#ifdef USE_CUDA

  __global__ void updateStateKernel(const float* const src,
                                    const int* const index,
                                    const UpdateStateParams params,
                                    float * const tgt) {
    // TODO(umiswing): niutensor doesn't guarantee 128-bit aligned.
    // Use 128-bit loading after 128-bit aligned support.
    constexpr int elements_per_thread = 1;
    const int threads_per_block = blockDim.x;
    const int elements_per_block = elements_per_thread * threads_per_block;
  
    const int src_hid = blockIdx.y;
    const int src_bid = index[blockIdx.x];
    const int src_hbid = (src_hid * params.src_batch_size + src_bid) * params.seqlen * params.head_dim;
  
    const int tgt_hid = blockIdx.y;
    const int tgt_bid = blockIdx.x;
    const int tgt_hbid = (tgt_hid * params.tgt_batch_size + tgt_bid) * params.seqlen * params.head_dim;
    #pragma unroll
    for(int i=threadIdx.x*elements_per_thread;i<params.seqlen * params.head_dim / elements_per_thread;i+=elements_per_block) {
      tgt[tgt_hbid+i] = src[src_hbid+i];
    }
  }

  void updateState(const XTensor* const src,
                   const XTensor* const index,
                   const struct UpdateStateParams params,
                   XTensor* const tgt) {
    CheckNTErrors(src != nullptr &&
                  index != nullptr &&
                  tgt != nullptr,
                  "Invalid tensor!");
    CheckNTErrors(src->dataType == X_FLOAT, "only support state with type X_FLOAT now!");
    CheckNTErrors(tgt->dataType == X_FLOAT, "only support state with type X_FLOAT now!");
    CheckNTErrors(index->dataType == X_INT, "index must be type X_INT!");
    CheckNTErrors(src->devID >= 0, "the state must be kept on the gpu!");
    CheckNTErrors(src->devID == tgt->devID, "the state must be kept on the same device!");
    CheckNTErrors((src->unitSize == tgt->unitSize), "Unmatched tensors!");

    int devID = src->devID;
    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    dim3 blocks(params.tgt_batch_size, params.num_head);

    dim3 threads(params.seqlen*params.head_dim > GDevs.GPUs[devID].GPUMaxThreadNumPerBlock ?
                 GDevs.GPUs[devID].GPUMaxThreadNumPerBlock :
                 params.seqlen*params.head_dim );

    updateStateKernel<<<blocks, threads>>>(static_cast<float*>(src->data),
                                           static_cast<int*>(index->data),
                                           params,
                                           static_cast<float*>(tgt->data));

    BacktoCudaDev(devID, devIDBackup);
  }
#endif // USE_CUDA

} /* end of the nmt namespace */